#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <float.h>

#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define MAX_THREADS 1024
#define NUM_REPETITIONS 5
#define TILE_SIZE 128

__global__ void matmul_kernel_tiled(float *A, float *B, float *C, int M, int N, int K) {
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;

    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        if (row < M && tile * TILE_SIZE + tx < K) {
            A_shared[ty][tx] = A[row * K + tile * TILE_SIZE + tx];
        } else {
            A_shared[ty][tx] = 0.0f;
        }

        if (col < N && tile * TILE_SIZE + ty < K) {
            B_shared[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
        } else {
            B_shared[ty][tx] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += A_shared[ty][k] * B_shared[k][tx];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

void matmul(float *A, float *B, float *C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    CHECK_CUDA_ERROR(hipMalloc(&d_A, size_A));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, size_B));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, size_C));

    CHECK_CUDA_ERROR(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));

    dim3 block_size(TILE_SIZE, TILE_SIZE);
    dim3 grid_size((N + TILE_SIZE - 1) / TILE_SIZE, 
                   (M + TILE_SIZE - 1) / TILE_SIZE);

    matmul_kernel_tiled<<<grid_size, block_size>>>(d_A, d_B, d_C, M, N, K);

    CHECK_CUDA_ERROR(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(int argc, char *argv[]) {
    int sizes[][3] = {{128, 128, 128}, {512, 512, 512}, {1024, 1024, 1024}, {2048, 2048,2048}, {4096, 4096, 4096}};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);
    int num_threads = 4;  // Default value
    if (argc > 1) {
        num_threads = atoi(argv[1]);
        if (num_threads <= 0 || num_threads > MAX_THREADS) {
            fprintf(stderr, "Invalid number of threads. Using default (4).\n");
            num_threads = 4;
        }
    }
    srand(time(NULL));
    
    printf("m,n,k,time,gflops\n");
    
    double best_gflops = 0.0;
    int best_m = 0, best_n = 0, best_k = 0;
    
    for (int i = 0; i < num_sizes; i++) {
        int M = sizes[i][0];
        int N = sizes[i][1];
        int K = sizes[i][2];
        float *A = (float*)malloc(M * K * sizeof(float));
        float *B = (float*)malloc(K * N * sizeof(float));
        float *C = (float*)malloc(M * N * sizeof(float));
        if (!A || !B || !C) {
            fprintf(stderr, "Memory allocation failed\n");
            exit(1);
        }
        
        for (int j = 0; j < M * K; j++) {
            A[j] = (float)rand() / RAND_MAX;
        }
        for (int j = 0; j < K * N; j++) {
            B[j] = (float)rand() / RAND_MAX;
        }
        
        double total_time = 0.0;
        double min_time = DBL_MAX;
        
        for (int rep = 0; rep < NUM_REPETITIONS; rep++) {
            double start_time = get_time();
            matmul(A, B, C, M, N, K);
            double end_time = get_time();
            double elapsed_time = end_time - start_time;
            
            total_time += elapsed_time;
            if (elapsed_time < min_time) {
                min_time = elapsed_time;
            }
        }
        
        double avg_time = total_time / NUM_REPETITIONS;
        double flops = 2.0 * M * N * K;
        double avg_gflops = flops / (avg_time * 1e9);
        double max_gflops = flops / (min_time * 1e9);
        
        printf("%d,%d,%d,%.6f,%.2f\n", M, N, K, avg_time, avg_gflops);
        
        if (max_gflops > best_gflops) {
            best_gflops = max_gflops;
            best_m = M;
            best_n = N;
            best_k = K;
        }
        
        free(A);
        free(B);
        free(C);
    }
    
    printf("\nBest configuration:\n");
    printf("M=%d, N=%d, K=%d\n", best_m, best_n, best_k);
    printf("Best performance: %.2f GFLOPS\n", best_gflops);
    
    return 0;
}
